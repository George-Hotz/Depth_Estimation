#include "hip/hip_runtime.h"
#include "preprocess.h"

static uint8_t *img_buffer_device = nullptr;

__global__ void warpaffine_kernel(
  uint8_t *src, int src_line_size, int src_width,
  int src_height, float *dst, int dst_width,
  int dst_height, uint8_t const_value_st,
  AffineMatrix d2s, int edge)
{
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= edge)
    return;

  float mean[] {0.485, 0.456, 0.406};
  float std[] {0.229, 0.224, 0.225};

  // 从d2s中读取变换矩阵
  float m_x1 = d2s.value[0];
  float m_y1 = d2s.value[1];
  float m_z1 = d2s.value[2];
  float m_x2 = d2s.value[3];
  float m_y2 = d2s.value[4];
  float m_z2 = d2s.value[5];

  int dx = position % dst_width; // 计算当前线程对应的目标图像的x坐标
  int dy = position / dst_width; // 计算当前线程对应的目标图像的y坐标

  float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
  float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
  float c0, c1, c2;

  if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height)
  {
    // 超出边界的像素点用const_value_st填充
    c0 = const_value_st;
    c1 = const_value_st;
    c2 = const_value_st;
  }else{
    // 双线性插值，实现图像的放大缩小
    int y_low = floorf(src_y);
    int x_low = floorf(src_x);
    int y_high = y_low + 1;
    int x_high = x_low + 1;

    uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
    float ly = src_y - y_low;
    float lx = src_x - x_low;
    float hy = 1 - ly;
    float hx = 1 - lx;
    float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    uint8_t *v1 = const_value;
    uint8_t *v2 = const_value;
    uint8_t *v3 = const_value;
    uint8_t *v4 = const_value;

    if (y_low >= 0)
    {
      if (x_low >= 0)
        v1 = src + y_low * src_line_size + x_low * 3;

      if (x_high < src_width)
        v2 = src + y_low * src_line_size + x_high * 3;
    }

    if (y_high < src_height)
    {
      if (x_low >= 0)
        v3 = src + y_high * src_line_size + x_low * 3;

      if (x_high < src_width)
        v4 = src + y_high * src_line_size + x_high * 3;
    }

    c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
    c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
    c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
  }

  // bgr to rgb
  float t = c2;
  c2 = c0;
  c0 = t;

  // normalization
  c0 = c0 / 255.0f;
  c1 = c1 / 255.0f;
  c2 = c2 / 255.0f;

  // imagenet normalization
  // c0 = (c0-mean[0])/std[0];
  // c1 = (c1-mean[1])/std[1];
  // c2 = (c2-mean[2])/std[2];

  // rgbrgbrgb to rrrgggbbb
  int area = dst_width * dst_height;
  float *pdst_c0 = dst + dy * dst_width + dx;
  float *pdst_c1 = pdst_c0 + area;
  float *pdst_c2 = pdst_c1 + area;
  *pdst_c0 = c0;
  *pdst_c1 = c1;
  *pdst_c2 = c2;
}


void cuda_preprocess(
    uint8_t *src, int src_width, int src_height,
    float *dst, int dst_width, int dst_height)
{

  int img_size = src_width * src_height * 3;
  CUDA_CHECK(hipMemcpy(img_buffer_device, src, img_size, hipMemcpyHostToDevice));

  // 计算变换矩阵
  AffineMatrix s2d, d2s;
  float scale = std::min(dst_height / (float)src_height, dst_width / (float)src_width);

  s2d.value[0] = scale;
  s2d.value[1] = 0;
  s2d.value[2] = -scale * src_width * 0.5 + dst_width * 0.5;
  s2d.value[3] = 0;
  s2d.value[4] = scale;
  s2d.value[5] = -scale * src_height * 0.5 + dst_height * 0.5;

  cv::Mat m2x3_s2d(2, 3, CV_32F, s2d.value);
  cv::Mat m2x3_d2s(2, 3, CV_32F, d2s.value);
  cv::invertAffineTransform(m2x3_s2d, m2x3_d2s);

  memcpy(d2s.value, m2x3_d2s.ptr<float>(0), sizeof(d2s.value));

  // 一个线程处理一个像素点，一共需要 dst_height * dst_width 个线程
  int jobs = dst_height * dst_width;
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  // 调用kernel函数

  warpaffine_kernel<<<blocks, threads>>>(
      img_buffer_device, src_width * 3, src_width,
      src_height, dst, dst_width,
      dst_height, 0, d2s, jobs);
}

void cuda_batch_preprocess(std::vector<cv::Mat> &img_batch,
                           float *dst, int dst_width, int dst_height)
{
  int dst_size = dst_width * dst_height * 3;
  for (size_t i = 0; i < img_batch.size(); i++)
  {
    cuda_preprocess(img_batch[i].ptr(), img_batch[i].cols, img_batch[i].rows, &dst[dst_size * i], dst_width, dst_height);
  }
}

void cuda_preprocess_init(int max_image_size)
{
  // prepare input data in device memory
  CUDA_CHECK(hipMalloc((void **)&img_buffer_device, max_image_size * 3));
}

void cuda_preprocess_destroy()
{
  CUDA_CHECK(hipFree(img_buffer_device));
}

// 使用cuda预处理所有步骤
void Preprocess_gpu(const cv::Mat &src, int inputW, int inputH, float *input_device_buffer)
{
  cuda_preprocess((uint8_t *)src.ptr(), src.cols, src.rows, input_device_buffer, inputW, inputH);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void Preprocess_cpu(const cv::Mat &img, int inputW, int inputH, void *input_host_buffer)
{
  cv::Mat resized;
  cv::resize(img, resized, cv::Size(inputW, inputH));
  cv::Mat rgb;
  cv::cvtColor(resized, rgb, cv::COLOR_BGR2RGB);
  cv::Mat normalized;
  rgb.convertTo(normalized, CV_32FC3);
  cv::subtract(normalized, cv::Scalar(127.5, 127.5, 127.5), normalized);
  cv::divide(normalized, cv::Scalar(127.5, 127.5, 127.5), normalized);
  // split it into three channels
  std::vector<cv::Mat> nchw_channels;
  cv::split(normalized, nchw_channels);

  for (auto &img : nchw_channels)
  {
      img = img.reshape(1, 1);
  }

  cv::Mat nchw;
  cv::hconcat(nchw_channels, nchw);

  memcpy(input_host_buffer, nchw.data, 3 * inputH * inputW * sizeof(float));
}